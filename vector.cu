#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__
void vecAddKernel(int * device)
{
  int i = threadIdx.x + (blockDim.x * blockIdx.x);
  
  
  
  device[i] = i;
  
}

void vecAdd()
{
  
  int inhost[100];
  int j;
  
  int * device;
  
  for( j = 0; j < 100; j++)
    inhost[j] = 0;
  
  for( j = 0; j < 100; j++)
    printf("inhost[%d] = %d\n", j, inhost[j]);
  
  printf("---------\n");
  
  hipMalloc(&device, 100*sizeof(int));

  // kernel invocation
  vecAddKernel<<<10,10>>>(device);

  //transfer C_d from device to host
  hipMemcpy(inhost, device, 400, hipMemcpyDeviceToHost);
  
  hipFree(device);
  
  for( j = 0; j < 100; j++)
    printf("inhost[%d] = %d\n", j, inhost[j]);

}


int main()
{
   vecAdd();
   
  
    return 0;
}
