#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define TOTAL_PRIMES 1000000

void start(unsigned long semiPrime);
void createPrimesArray(unsigned int *array);

int main ( int argc, char *argv[] ) {
    // Read in file
    if (argc < 2) {
        printf("Sorry, we need a command line argument\n"); 
        printf("Run again with Semiprime you would like to factor\n");
        exit(0);
    }

    else {
        char* semiPrime = argv[1];
        char* ptr;
        unsigned long longSemiPrime;
        longSemiPrime = strtoul(semiPrime, &ptr, 10);
        start(longSemiPrime); 
    }
}

void start(unsigned long semiPrime) {
    //unsigned long *primes;
    //primes = malloc(sizeof(unsigned long) * TOTAL_PRIMES);
    
    unsigned int primes[TOTAL_PRIMES];
    createPrimesArray(&primes[0]);
    printf("Finished Extracting Primes\n");

    unsigned int * d_primes;
    unsigned int * d_solution;

    // calculate grid value
    int totalBlocks = (TOTAL_PRIMES / 512) + 1;
    int gridDimensions = sqrt(totalBlocks) + 1;

    dim3 dimGrid(gridDimensions, gridDimensions, 1);
    dim3 dimBlock(16, 16, 1);

    // Allocate host memory
    unsigned int solution[2];
    for (int i = 0; i < 2; i++) {
        solution[i] = 0;
    }

    hipMalloc(&d_primes, TOTAL_PRIMES *sizeof(unsigned int));
    hipMalloc(&d_solution, 2*sizeof(unsigned int));
    
    // Kernel Invocation
    vecAddKernel<<<dimGrid,dimBlock>>>(device);

    //transfer C_d from device to host
    hipMemcpy(solution, d_solution, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);
  
    hipFree(d_solution);
    hipFree(d_primes);

    int totalPrimes = TOTAL_PRIMES;
    if (solution[0] == 0 && solution[1] == 0) {
        printf("%lu is not a semiprime with factors less than %d\n", semiPrime, totalPrimes); 
    }

    else {
        printf("The factors of %lu are %u and %u\n", semiPrime, solution[0], solution[1]);
    }
}

void createPrimesArray(unsigned int *array) {
    FILE *inputFile;
    unsigned int mostRecentNumber;
    inputFile = fopen("primes1.txt", "r");
    for (int i = 0; i < TOTAL_PRIMES; i++) {
        fscanf(inputFile, "%u", &mostRecentNumber);
        array[i] = mostRecentNumber;
    }
}

__global__ 
void simpleAssign(int* d_array,int* d_solution, unsigned long semiPrime) {
    d_solution[0] = 69;
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y
    unsigned int xValue = d_primes[xIndex];
    unsigned int yValue = d_primes[yIndex];

    unsigned long value = (unsigned long)xValue * (unsigned long)yValue;

    if (value == semiPrime) {
        d_solution[0] = xValue;
        d_solution[1] = yValue;
    }
}
