
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define TOTAL_PRIMES 1000000

void start(unsigned long semiPrime);
void createPrimesArray(unsigned int *array);
__global__ void factorSemiprime(unsigned int* d_primes, unsigned int* d_solution, unsigned long semiPrime);

int main ( int argc, char *argv[] ) {
    // Read in file
    if (argc < 2) {
        printf("Sorry, we need a command line argument\n"); 
        printf("Run again with Semiprime you would like to factor\n");
        exit(0);
    }

    else {
        char* semiPrime = argv[1];
        char* ptr;
        unsigned long longSemiPrime;
        longSemiPrime = strtoul(semiPrime, &ptr, 10);
        start(longSemiPrime); 
    }
}

void start(unsigned long semiPrime) {
    //unsigned long *primes;
    //primes = malloc(sizeof(unsigned long) * TOTAL_PRIMES);
    
    printf("\n**************Starting GPU***************\n"); 
    // Initialize primes
    unsigned int primes[TOTAL_PRIMES];
    for (int i = 0; i < TOTAL_PRIMES; i++) {
        primes[i] = 0;
    }

    createPrimesArray(&primes[0]);

    // Allocate host memory
    unsigned int *solution = (unsigned int*) malloc(sizeof(unsigned int) * 2);

    //unsigned int solution[2];
    for (int i = 0; i < 2; i++) {
        solution[i] = 0;
    }

    unsigned int * d_primes;
    unsigned int * d_solution;

    // calculate grid value
    int gridDimensions = (TOTAL_PRIMES / 16) + 1;

    //int totalBlocks = (TOTAL_PRIMES / 512) + 1;
    //int gridDimensions = sqrt(totalBlocks) + 1;
    //printf("The Grid dimension is %d x %d\n", gridDimensions, gridDimensions);

    dim3 dimGrid(gridDimensions, gridDimensions, 1);
    dim3 dimBlock(16, 16, 1);

    hipMalloc((void**)&d_primes, TOTAL_PRIMES *sizeof(unsigned int));
    hipMalloc((void**)&d_solution, 2 * sizeof(unsigned int));

    // Copy primes to GPU
    hipMemcpy(d_primes, primes, TOTAL_PRIMES * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_solution, solution, 2 * sizeof(unsigned int), hipMemcpyHostToDevice);
    
    // Kernel Invocation
    factorSemiprime<<<dimGrid,dimBlock>>>(d_primes, d_solution, semiPrime);

    //transfer C_d from device to host
    hipMemcpy(solution, d_solution, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

    int totalPrimes = TOTAL_PRIMES;
    if (solution[0] == 0 && solution[1] == 0) {
        printf("%lu is not a semiprime with factors less than %d\n\n", semiPrime, totalPrimes); 
    }

    else {
        printf("The prime factors of %lu are %u and %u\n\n", semiPrime, solution[0], solution[1]);
    }

    hipFree(d_solution);
    hipFree(d_primes);
    free(solution);
}

void createPrimesArray(unsigned int *array) {
    FILE *inputFile;
    unsigned int mostRecentNumber;
    inputFile = fopen("primes1.txt", "r");
    for (int i = 0; i < TOTAL_PRIMES; i++) {
        fscanf(inputFile, "%u", &mostRecentNumber);
        array[i] = mostRecentNumber;
    }
}

__global__ 
void factorSemiprime(unsigned int* d_primes, unsigned int* d_solution, unsigned long semiPrime) {
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    // Exit on the edge threads that exceed our data
    if (xIndex > TOTAL_PRIMES || yIndex > TOTAL_PRIMES) {
        return;
    }

    unsigned int xValue = d_primes[xIndex];
    unsigned int yValue = d_primes[yIndex];

    unsigned long value = (unsigned long)xValue * (unsigned long)yValue;

    if (value == semiPrime) {
        d_solution[0] = xValue;
        d_solution[1] = yValue;
    }
}
