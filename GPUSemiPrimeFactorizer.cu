#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define TOTALNUMBERS 100
void startSeq(int number);

int main (int argc, char* argv[]) {
    startSeq(100);
}

__global__ 
void simpleAssign(int* d_solution) {
    d_solution[0] = 69;
}

void startSeq(int number) {

    //Read File
    int *inputPrimes;
    
    dim3 dimGrid(10, 1, 1);
    dim3 dimBlock(10, 1, 1);

    int *d_solution;
    int *solution;

    solution = (int *) malloc(TOTALNUMBERS * sizeof(int));

    for (int i = 0; i < TOTALNUMBERS; i++) {
        solution[i] = 0;
    }

    for (int i = 0; i < TOTALNUMBERS; i++) {
        printf("%d: Before GPU is %d", i, solution[i]);
    }

	//cudaHostAlloc((void**)&solution,sizeof(int),cudaHostAllocDefault);
    //solution[0] = 10;

    //printf("The solution before is %d\n", solution[0]);

    hipMalloc(&d_solution, sizeof(int) * TOTALNUMBERS);
    simpleAssign<<<dimGrid,dimBlock>>>(d_solution);
    hipMemcpy(solution, d_solution, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_solution);

    for (int i = 0; i < TOTALNUMBERS; i++) {
        printf("%d: After GPU is %d", i, solution[i]);
    }
}
